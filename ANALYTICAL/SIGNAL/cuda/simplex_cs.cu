#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <fstream>
#include <stdio.h>
#include <random>
//This program simulates restricted diffusion with exchange in a substrate of 
//analytically defined parallel cylinders
//It computes signals at a resolution saved by the user.

//Author: 
//Arthur Chakwizira, Medical Radiation Physics, Lund University, Sweden
//arthur.chakwizira@med.lu.se


//simulation options; will be read from an options txt file
struct options
{
	long Npart;
	float T;
	float sim_dt;
	bool do_samp;
	float samp_dt;
	int n_dim;
	float D0_intra, D0_extra;
	long sim_Nt;
	long save_Nt;
	float ds_intra, ds_extra;
	long N_save; //N time points x N particles
	long N_sim;
	bool save_states; //save particle state history to file or not
	float kappa, p_12, p_21; //membrane permeability
	//option to initialise all spins in one compartment and allow one-directional transitions
	bool all_intra, all_extra, intra_to_extra_allowed, extra_to_intra_allowed;
	//additional gwf options
	long n_acq = 0; //# waveforms
	long n_gwf_points = 0; // # time points in each waveform
	float gamma; //gyromagnetic ratio
	long delay; //number of time steps to take before acquiring signals
	bool save_positions; //save final positions or not; useful for debugging
	float gwf_dt; //gradient waveform resolution
	bool save_phase; //save phase or not, might be useful for computing cumulants
};


//substrate info
struct world
{
	long long num_cells, num_voxels;
	float max_x, max_y, max_z, x_length, y_length, z_length, f1, vox_size;
};

//initialise rng with seeds based on system clock
__global__ void random_init(hiprandState* states)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(clock64()+index, index, 0, &states[index]);
}

//function for pairing particle coordinates using the Szudzik pairing algorithm 
__device__ void pair(long long x, long long y, long long& xy)
{
	//device function for pairing particle coordinates
	x >= 0 ? x = 2 * x : x = -2 * x - 1;
	y >= 0 ? y = 2 * y : y = -2 * y - 1;

	x >= y ? xy = x * x + x + y : xy = y * y + x;
}

//binary search to find index of element in array
__device__ long long binary_search_iter(long long* A, long long lower, long long upper, long long x)
{
	while (upper >= lower) {
		long long mid = lower + (upper - lower) / 2;
		if (A[mid] == x) return mid;
		(A[mid] > x) ? upper = mid - 1 : lower = mid + 1;
	}
	return -1;
}

//determine if particle is inside any cylinder in the substrate
__device__ bool is_particle_in_any_cell(float tmp_x, float tmp_y, world* w, float* centre_x, float* centre_y, float* radii, long long* table, long long* cell_idx)
{
	long long x_pos = floor(tmp_x / w->vox_size);
	long long y_pos = floor(tmp_y / w->vox_size);
	long long xy; //voxel identifier
	long long which_cell, which_voxel;
	bool inside = false; //zero means outside all cells
	float distance, r, cx, cy;
	
	pair(x_pos, y_pos, xy); //get the identifier

	which_voxel = binary_search_iter(table, 0, w->num_voxels - 1, xy); //iterative binary search	
	
	which_cell = cell_idx[which_voxel]-1; //index of cell containing the voxel containing the particle
	//note the minus 1 to take into account that MATLAB numbering starts at 1. Need to fix this later
    
    r = radii[which_cell];
    cx = centre_x[which_cell];
    cy = centre_y[which_cell];
	
	if (which_cell >= 0) //-1 means voxel is not in any cell
	{
		distance = (tmp_x-cx)*(tmp_x-cx) + (tmp_y-cy)*(tmp_y-cy);
		if ( distance <= (r*r)  ) //means particle is in this cell
		{
			inside = true;
		}
	}
	return inside;
}


//make diffusion step; allows for different intra/extra diffusivities
__device__ void diffuse(float& tmp_x, float& tmp_y, float& tmp_z, float& tmp_dx,
	float& tmp_dy, float& tmp_dz, hiprandState* states, int index, options* opt, int tmp_loc)
{
	tmp_dx = hiprand_normal(&states[index]);
	tmp_dy = hiprand_normal(&states[index]);
	tmp_dz = hiprand_normal(&states[index]);

	float norm;
	if (tmp_loc == 1) //particle is in mesh
	{
		norm = (*opt).ds_intra * rnorm3df(tmp_dx, tmp_dy, tmp_dz);
	}
	else  //particle is not in mesh
	{
		norm = (*opt).ds_extra * rnorm3df(tmp_dx, tmp_dy, tmp_dz);
	}
	
	tmp_dx *= norm;
	tmp_dy *= norm;
	tmp_dz *= norm;

	tmp_x += tmp_dx;
	tmp_y += tmp_dy;
	tmp_z += tmp_dz;
}


//implementation of periodic boundary conditions
__device__ void restrict_to_world(float& e_x, float& e_y, float& e_z, options* opt, world* w, float& tmp_x, float& tmp_y, float& tmp_z)
{
	if (tmp_x < -w->max_x) { tmp_x += w->x_length; e_x -= w->x_length; }
	if (tmp_x >= w->max_x) { tmp_x -= w->x_length; e_x += w->x_length; }

	if (tmp_y < -w->max_y) { tmp_y += w->y_length; e_y -= w->y_length; }
	if (tmp_y >= w->max_y) { tmp_y -= w->y_length; e_y += w->y_length; }

	if (tmp_z < -w->max_z) { tmp_z += w->z_length; e_z -= w->z_length; }
	if (tmp_z >= w->max_z) { tmp_z -= w->z_length; e_z += w->z_length; }
}

//membrane intersection checks and exchange mechanism
__device__ void check_state(world* w, options* opt, float& tmp_x, float& tmp_y, float& tmp_z, float& tmp_dx,
	float& tmp_dy, float& tmp_dz, int& tmp_loc, float* centre_x, float* centre_y, float *radii, long long* table, long long * cell_idx,  hiprandState* states, int index)
{
	bool reject = false; //reject move or not
	bool inside;

	inside = is_particle_in_any_cell(tmp_x, tmp_y, w, centre_x, centre_y, radii, table, cell_idx); //determine whether particle is in any cell

	if (inside) //"now intra"
	{
		if (tmp_loc == 1) { reject = false; } //was intra before
		else //was not intra before
		{
			if (hiprand_uniform(&states[index]) < opt->p_21) { reject = false; tmp_loc = 1; }
			else { reject = true; tmp_loc = 0; }
		}
	}

	if (!inside) //"now extra"
	{
		if (tmp_loc == 0) { reject = false; } //was extra before
		else //was not extra before
		{
			if (hiprand_uniform(&states[index]) < opt->p_12) { reject = false; tmp_loc = 0; }
			else { reject = true; tmp_loc = 1; }
		}
	}


	if (reject) { tmp_x -= tmp_dx; tmp_y -= tmp_dy; tmp_z -= tmp_dz; }

}


__global__ //this kernel runs the actual simulation
void engine(float* x, float* y, float* z, int* loc, float* centre_x, float* centre_y, float* radii, long long* table, long long* cell_idx,
	hiprandState* states, options* opt, world* w, float* phase, float * gwf_x, float *gwf_y, float* gwf_z)
{
	long index = blockIdx.x * blockDim.x + threadIdx.x; //thread index
	//int stride = blockDim.x * gridDim.x;

	long phase_entry;
	long save;
	long save_c_t;
	float tmp_x, tmp_y, tmp_z;
	float tmp_dx, tmp_dy, tmp_dz;
	float e_x, e_y, e_z; //keep track of hyperposition
	int tmp_loc;
	long sampling_interval = (long) std::round(opt->samp_dt / opt->sim_dt);

	long c_p = index;
	if (c_p < opt->Npart) //ensure we keep within bounds
	{
		tmp_x = x[c_p];
		tmp_y = y[c_p];
		tmp_z = z[c_p];
		tmp_dx = 0;
		tmp_dy = 0;
		tmp_dz = 0;
		tmp_loc = loc[c_p];
		
		if (index == opt->Npart-1) {printf("Preparing simulation loop ... \n");}
		
		//adding a delay loop here
		e_x = 0; e_y = 0; e_z = 0;
		for (int c_t = 0; c_t < 50; c_t++)
		{
			diffuse(tmp_x, tmp_y, tmp_z, tmp_dx, //make diffusion steps
				tmp_dy, tmp_dz, states, index, opt, tmp_loc);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
			check_state(w, opt, tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, tmp_loc, centre_x, centre_y, radii, table, cell_idx, states, index);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
		}

		if (index == opt->Npart-1) {printf("Starting simulation loop ... \n");}
		
		e_x = 0; e_y = 0; e_z = 0;
		save_c_t = -1;
		save = 0;
		for (long c_t = 0; c_t < (*opt).sim_Nt; c_t++)
		{
			
			diffuse(tmp_x, tmp_y, tmp_z, tmp_dx, //make diffusion steps
				tmp_dy, tmp_dz, states, index, opt, tmp_loc);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
			check_state(w, opt, tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, tmp_loc, centre_x, centre_y, radii, table, cell_idx, states, index);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
		
			save++;

			if (save == sampling_interval)
			{
				save_c_t++;
				for (long c_a = 0; c_a < opt->n_acq; c_a++)
					{
						phase_entry = c_p * (*opt).n_acq + c_a;
						phase[phase_entry] += opt->gamma*((tmp_x+e_x)*gwf_x[c_a*(*opt).save_Nt+save_c_t] + (tmp_y+e_y)*gwf_y[c_a*(*opt).save_Nt+save_c_t] + (tmp_z+e_z)*gwf_z[c_a*(*opt).save_Nt+save_c_t])*opt->samp_dt;
					}
					save = 0;
				if (index==opt->Npart-1 && save_c_t%100==0){printf("Step %d of %ld \n", save_c_t, (*opt).save_Nt);}
			}
		}
		if (index == opt->Npart-1) {printf("Finishing simulation loop ... \n");}
	//save last position and state
	x[c_p] = tmp_x+e_x; y[c_p] = tmp_y+e_y; z[c_p] = tmp_z+e_z; loc[c_p] = tmp_loc;
	}
}


bool set_options(options* opt, char* pos_fn, char* sta_fn, char* sub_fn, char* gwf_fn,
char* sig_fn, char* phase_fn)
{
	//sets options loaded from file
	std::string opt_fn = ".\\options.txt";
	std::string dummy;

	std::ifstream tf;
	tf.open(opt_fn, std::ios::in);
	if (tf.is_open())
	{
	tf >> dummy >> opt->Npart;
	tf >> dummy >> opt->sim_dt;
	tf >> dummy >> opt->D0_intra;
	tf >> dummy >> opt->D0_extra;
	tf >> dummy >> opt->kappa;
	tf >> dummy >> opt->all_intra;
	tf >> dummy >> opt->all_extra;
	tf >> dummy >> opt->intra_to_extra_allowed;
	tf >> dummy >> opt->extra_to_intra_allowed;
	tf >> dummy >> sub_fn;
	tf >> dummy >> gwf_fn;
	tf >> dummy >> sig_fn;
	tf >> dummy >> opt->save_positions;
	tf >> dummy >> pos_fn;
	tf >> dummy >> opt->save_states;
	tf >> dummy >> sta_fn;
	tf >> dummy >> opt->save_phase;
	tf >> dummy >> phase_fn;
	tf.close();


	opt->n_dim = 3;
	opt->gamma = 2.675129e8;
	
	opt->ds_extra = (float)sqrt(2 * (*opt).n_dim * (*opt).D0_extra * (*opt).sim_dt); //diffusion step size extra
	opt->ds_intra = (float)sqrt(2 * (*opt).n_dim * (*opt).D0_intra * (*opt).sim_dt); //diffusion step size intra

	//Add transition probablities here //Lee-2021, Eq.3
	//double Cd = 2.0/3, k = opt->kappa, ds1 = (opt->ds_blood), D1 = opt->D0_blood;
	//double ds2 = (opt->ds_extra), D2 = opt->D0_extra;
	//opt->p_12 = (k*ds1*Cd/D1) / (1 + (k/2)*( (ds1/D1)  + (ds2/D2) )*Cd);
	//opt->p_21 = (k*ds2*Cd/D2) / (1 + (k/2)*( (ds2/D2)  + (ds1/D1) )*Cd);

	opt->p_21 = (float)opt->kappa * sqrt(8 * opt->sim_dt / (3 * opt->D0_extra)); 
	if (!opt->extra_to_intra_allowed) {opt->p_21 = 0;};
	opt->p_12 = (float)opt->kappa * sqrt(8 * opt->sim_dt / (3 * opt->D0_intra));
	if (!opt->intra_to_extra_allowed) {opt->p_12 = 0;};
	
	std::cout << "Loaded options from: " << opt_fn << std::endl;
	std::cout << "Substrate filename is : " << sub_fn << std::endl;
	std::cout << "Simulation time step =  " << opt->sim_dt << std::endl;
	std::cout << "Intra diffusivity = " << opt->D0_intra << std::endl;
	std::cout << "Membrane permeability = " << opt->kappa << std::endl;
	std::cout << "Transition probability intra->extra : " << opt->p_12 << std::endl;
	std::cout << "Transition probability extra->intra : " << opt->p_21 << std::endl;
	return true;
	}
	else
	{std::cout << "\n ERROR loading options from: " << opt_fn << " \n" << std::endl;
		return false;
	}
}

//saves trajectories
void save_trajectory(float* x, float* y, float* z, char* r_fn, options* opt)
{
	long save_Nt = 1;
	std::cout << "Saving final position to: " << r_fn << std::endl;
	FILE* tf;
	tf = fopen(r_fn, "wb");
	if (tf)
	{
	fwrite(&(opt->Npart), sizeof(long), 1, tf);
	fwrite(&(opt->T), sizeof(float), 1, tf);
	fwrite(&(save_Nt), sizeof(long), 1, tf);
	fwrite(x, sizeof(float), opt->Npart, tf); 
	fwrite(y, sizeof(float), opt->Npart, tf);
	fwrite(z, sizeof(float), opt->Npart, tf);
	fclose(tf);
	}
	else
	{
		std::cout << "\n ERROR: Failed to save final positions to: " << r_fn << std::endl;
	}
}

//saves history of particle identities/compartment identities
void save_state_history(int* s, char* s_fn, options* opt)
{
	std::cout << "Saving transition history to: " << s_fn << std::endl;
	FILE* tf;
	tf = fopen(s_fn, "wb");
	if (tf)
	{
	fwrite(&(opt->Npart), sizeof(long), 1, tf);
	fwrite(&(opt->T), sizeof(float), 1, tf);
	fwrite(&(opt->save_Nt), sizeof(long), 1, tf);
	fwrite(s, sizeof(int), opt->N_save, tf);
	fclose(tf);
	}
	else
	{
		std::cout << "\n ERROR: Failed to save transition history to: " << s_fn << std::endl;
	}
}

//save signal to file
void save_signal(float* h_signal, char* sig_fn, options* opt)
{
	
	std::cout << "Saving signals to: " << sig_fn << std::endl;
	FILE* tf;
	tf = fopen(sig_fn, "wb");
	if (tf)
	{
	fwrite(h_signal, sizeof(float), opt->n_acq, tf);
	fclose(tf);
	}
	else
	{
		std::cout << "\n ERROR: Failed to save signals to: " << sig_fn << std::endl;
	}
}

//open substrate file and get number of voxels
bool get_substrate_info(options* opt, world* w, char* g_fn)
{
	FILE* sf;
	sf = fopen(g_fn, "rb");
	if (sf)
	{
	fread(&(w->num_cells), sizeof(long long), 1, sf);
	fread(&(w->num_voxels), sizeof(long long), 1, sf);
	fread(&(w->max_x), sizeof(float), 1, sf);
	fread(&(w->max_y), sizeof(float), 1, sf);
	fread(&(w->max_z), sizeof(float), 1, sf);
	fread(&(w->vox_size), sizeof(float), 1, sf);
	fclose(sf);
	if (w->vox_size < std::max(opt->ds_intra, opt->ds_extra)){
		std::cout << "\n ERROR: Substrate voxel size of " << w->vox_size << 
		" is smaller than the simulation step-size of " << std::max(opt->ds_intra, opt->ds_extra) <<std::endl;
		return false;
	}
	else{
	return true;}
	}
	else 
	{std::cout << "\n ERROR: Specified substrate file: " << g_fn << "  does not exist! \n" <<std::endl;
		return false;}
}

//open gwf file and get info about the gwf
bool get_gwf_info(char* gwf_fn, options* opt)
{
	std::cout << "Getting gradient waveform info from: " << gwf_fn << std::endl;
	FILE* sf;
	sf = fopen(gwf_fn, "rb");
	if (sf)
	{
	fread(&(opt->n_acq), sizeof(long), 1, sf);
	fread(&(opt->n_gwf_points), sizeof(long), 1, sf);
	fread(&(opt->gwf_dt), sizeof(float), 1, sf);
	fclose(sf);
	
	//sampling time step is equal to gwf time-step
	opt->samp_dt = opt->gwf_dt;
	if (opt->samp_dt < opt->sim_dt){
		std::cout << "\n ERROR: Simulation time-step of " << opt->sim_dt << "is larger than the gradient waveform resolution of " << opt->samp_dt <<std::endl;
		return false;
	}
	else{
	//total simulation time is number of points in gwf*gwf_dt
	opt->sim_Nt = opt->n_gwf_points * (opt->samp_dt/opt->sim_dt) +1; //total simulation time steps
	//set this also in variable T
	opt->T = (opt->sim_Nt)*opt->sim_dt;
	
	opt->save_Nt = opt->n_gwf_points;//(long long)round(opt->T / opt->samp_dt);
	opt->N_save = (long)opt->Npart * opt->save_Nt; //N time points x N particles
	opt->N_sim = (long)opt->Npart * opt->sim_Nt;
	return true;
	}
	}
	else
	{
		std::cout << "\n ERROR: Specified gradient waveform file: " << gwf_fn << "  does not exist! \n" <<std::endl;
		return false;
	}
}

//load simulation substrate from file
bool load_substrate(float* h_centre_x, float* h_centre_y, float* h_radii, long long* h_table, long long *h_cell_idx, world* w, char* g_fn)
{
	FILE* sf;
	sf = fopen(g_fn, "rb");
	if (sf)
	{
	fread(&(w->num_cells), sizeof(long long), 1, sf);
	fread(&(w->num_voxels), sizeof(long long), 1, sf);
	fread(&(w->max_x), sizeof(float), 1, sf);
	fread(&(w->max_y), sizeof(float), 1, sf);
	fread(&(w->max_z), sizeof(float), 1, sf);
	fread(&(w->vox_size), sizeof(float), 1, sf);
	fread(&w->f1, sizeof(float), 1, sf);
	fread(h_centre_x, sizeof(float), w->num_cells, sf); 
	fread(h_centre_y, sizeof(float), w->num_cells, sf); 
	fread(h_radii, sizeof(float), w->num_cells, sf); 
	fread(h_table, sizeof(long long), w->num_voxels, sf); 
	fread(h_cell_idx, sizeof(long long), w->num_voxels, sf); 
	fclose(sf);

	w->x_length = 2 * w->max_x;
	w->y_length = 2 * w->max_y;
	w->z_length = 2 * w->max_z;

	std::cout << "Loaded substrate from: " << g_fn << std::endl;
	std::cout << "Intra volume fraction =  " <<  w->f1 << std::endl;
	std::cout << "Number of voxels =  " <<  w->num_voxels << std::endl;
	std::cout << "Voxel size =  " <<  w->vox_size << std::endl;
	return true;
	}
	{
	std::cout << "\n ERROR reading substrate from:  " << g_fn << " \n" << std::endl;		
	return false;
	}
}

//load gradient waveforms from file
bool load_gwf(char* gwf_fn, options* opt, float* h_gwf_x, float* h_gwf_y, float* h_gwf_z)
{
	std::cout << "Loading gradient waveform..."<< std::endl;
	//load substrate from file
	FILE* sf;
	sf = fopen(gwf_fn, "rb");
	if (sf)
	{
	fread(&(opt->n_acq), sizeof(long), 1, sf);
	fread(&(opt->n_gwf_points), sizeof(long), 1, sf);
	fread(&(opt->gwf_dt), sizeof(float), 1, sf);
	fread(h_gwf_x, sizeof(float), opt->n_acq*opt->n_gwf_points, sf);
	fread(h_gwf_y, sizeof(float), opt->n_acq*opt->n_gwf_points, sf);
	fread(h_gwf_z, sizeof(float), opt->n_acq*opt->n_gwf_points, sf); 
	fclose(sf);
	return true;
	}
	else
	{
		std::cout << "\n ERROR loading gradient waveforms from:  " << gwf_fn << " \n" << std::endl;
		return false;
	}

}

//convert phase to signal
void convert_phase_to_signal(float *h_phase, float *h_signal, options *opt)
{
	std::cout << "Converting phase to signal..." << std::endl;
	float sum_cos_phase;
	long phase_entry;
	for (long ca = 0; ca<opt->n_acq; ca++)
	{
		sum_cos_phase = 0;
	for (long c_p = 0; c_p < opt->Npart; c_p++)
		{
			phase_entry = c_p * (*opt).n_acq + ca;
			sum_cos_phase += cos(h_phase[phase_entry]);
		}
		h_signal[ca] = sum_cos_phase/opt->Npart;
	}
	 std::cout << "Done." << std::endl;
}

//save phase to file
void save_phase(float* h_phase, options *opt, char* sta_fn)
{
		
	long N = opt->Npart*opt->n_acq;
	std::cout << "Saving phase to: " << sta_fn << std::endl;
	FILE* tf;
	tf = fopen(sta_fn, "wb");
	if (tf)
	{
	fwrite(&N, sizeof(long), 1, tf);
	fwrite(h_phase, sizeof(float), opt->n_acq*opt->Npart, tf);
	fclose(tf);
	std::cout << "Done." << std::endl;
	}
	else
	{
		std::cout << "\n ERROR saving phase to: " << sta_fn << std::endl;
	}
}



//generate starting positions for all particles
__global__ void generate_initial_distribution(float* x, float* y, float* z, int* loc, float* centre_x,
	float* centre_y, float* radii, long long* table, long long* cell_idx, hiprandState* states, options* opt, world* w)
{
	long index = blockIdx.x * blockDim.x + threadIdx.x; //get thread idx
	float tmp_x, tmp_y, tmp_z; //frac defines intra-extra split of initial populations
	long idx;
	bool inside;
	bool success = false;
	float intra_frac; //fraction of spins to put intra
	if (opt->all_intra && !opt->all_extra)  {intra_frac = 1;};
	if (opt->all_extra && !opt->all_intra)  {intra_frac = 0;};
	if (opt->all_intra && opt->all_extra)  {intra_frac = 0.5;}; //if both are true, divide by half
	if (!opt->all_intra && !opt->all_extra)  {intra_frac = w->f1;};
	
	if (index < opt->Npart)
	{
		//places particles in initial positions all over substrate

		if (hiprand_uniform(&states[index]) < intra_frac) //intra
		{
			while (!success)
			{
			tmp_x = -w->max_x + 2 * hiprand_uniform(&states[index]) * w->max_x; //suggest initial position
			tmp_y = -w->max_y + 2 * hiprand_uniform(&states[index]) * w->max_y; //suggest initial position
			tmp_z = -w->max_z + 2 * hiprand_uniform(&states[index]) * w->max_z; //suggest initial position
			
			inside = is_particle_in_any_cell(tmp_x, tmp_y, w, centre_x, centre_y, radii, table, cell_idx); //determine whether particle is in any cell
			
			if (inside)
			{
				idx = index;
				x[idx] = tmp_x;
				y[idx] = tmp_y;
				z[idx] = tmp_z;
				loc[idx] = 1;
				success = true;
			}
			}
		}
		else
		{
			while (!success)
			{
			tmp_x = -w->max_x + 2 * hiprand_uniform(&states[index]) * w->max_x; //suggest initial position
			tmp_y = -w->max_y + 2 * hiprand_uniform(&states[index]) * w->max_y; //suggest initial position
			tmp_z = -w->max_z + 2 * hiprand_uniform(&states[index]) * w->max_z; //suggest initial position
			
			inside = is_particle_in_any_cell(tmp_x, tmp_y, w, centre_x, centre_y, radii, table, cell_idx); //determine whether particle is in any cell
			
			if (!inside)
			{
				idx = index;
				x[idx] = tmp_x;
				y[idx] = tmp_y;
				z[idx] = tmp_z;
				loc[idx] = 0;
				success = true;
			}
			}
		}
	}
}



int main(void)
{
	//OPENING STATEMENTS
	//__________________________________________________________________
	std::clock_t start;
	float duration;
	start = std::clock();
	hipError_t error = hipSuccess;
	int nDevices;
	hipGetDeviceCount(&nDevices);
	printf("\n PARALLEL SIMULATIONS OF DIFFUSION WITH EXCHANGE (SIMPLEX) \n");
	printf(" --- Hardware information ---\n");
	printf("Number of GPUs: %d\n", nDevices);
	int activeDevice;
	hipGetDevice(&activeDevice);
	printf("Active GPU index: %d\n", activeDevice);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, activeDevice);
	printf("GPU name: %s\n", prop.name);
	printf(" --------------------  \n");
	//__________________________________________________________________
	
	//DECLARATIONS
	//__________________________________________________________________
	bool PROCEED = true; //If false, simulation will abort
	//non-numerical options, host-only
	char pos_fn[500], sta_fn[500], sub_fn[500], gwf_fn[500], sig_fn[500], phase_fn[500]; //filenames for trajectories, states, substrate, gradient, signal and phase
	//options structure for host
	options* opt, * dev_opt;
	//declare gradient waveform arrays
	float *h_gwf_x, *h_gwf_y, *h_gwf_z; //waveform in x y z
	float *dev_gwf_x, *dev_gwf_y, *dev_gwf_z;
	//declare substrate structure for device and host
	world* h_w, * dev_w;
	//declare substrate arrays on device and host
	float* h_centre_x, *h_centre_y, *h_radii;
	float* dev_centre_x, *dev_centre_y, *dev_radii;
	long long *h_table, *h_cell_idx;
	long long *dev_table, *dev_cell_idx;
	//declare traj arrays and particle location (compartment id)
	int* h_loc, * dev_loc;
	float* h_x, * h_y, * h_z; //for the host
	float* dev_x, * dev_y, * dev_z; //for the device
	//cuRand states
	hiprandState* dev_states;
	//declare signal and phase arrays
	float* h_signal, *h_phase; //for the host
	float * dev_phase; //for the device
	//__________________________________________________________________
	
	//OPTIONS
	//__________________________________________________________________
	opt = (options*)malloc(sizeof(options));
	PROCEED = PROCEED && set_options(opt, pos_fn, sta_fn, sub_fn, gwf_fn, sig_fn, phase_fn);
	if (PROCEED){
	error = hipMalloc(&dev_opt, sizeof(options));
	if (error != hipSuccess){
		std::cout << "\n ERROR: FAILED TO ALLOCATE OPTIONS MEMORY ON GPU! \n" << std::endl;
		PROCEED = false;}
	}
	//__________________________________________________________________
	
	//GRADIENT WAVEFORMS
	//__________________________________________________________________
	//get info about gradient waveforms (how many and how many time points in each)
	if (PROCEED){
		PROCEED = get_gwf_info(gwf_fn, opt);}
		
	if (PROCEED){
	//allocate gradient waveform memory on host
	h_gwf_x = (float*)malloc(opt->n_acq*opt->n_gwf_points* sizeof(float));
	h_gwf_y = (float*)malloc(opt->n_acq*opt->n_gwf_points* sizeof(float));
	h_gwf_z = (float*)malloc(opt->n_acq*opt->n_gwf_points* sizeof(float));
	//load gradient waveform
	PROCEED = load_gwf(gwf_fn, opt, h_gwf_x, h_gwf_y, h_gwf_z);}
	if (PROCEED){
	//allocate gradient waveform memory on device
	hipMalloc(&dev_gwf_x, opt->n_acq*opt->n_gwf_points * sizeof(float));
	hipMalloc(&dev_gwf_y, opt->n_acq*opt->n_gwf_points * sizeof(float));
	error = hipMalloc(&dev_gwf_z, opt->n_acq*opt->n_gwf_points * sizeof(float));
	if (error != hipSuccess){
		std::cout << "\n ERROR: FAILED TO ALLOCATE GRADIENT WAVEFORM MEMORY ON GPU! \n" << std::endl;
		PROCEED = false;}
	else{
		hipMemcpy(dev_gwf_x, h_gwf_x, opt->n_acq*opt->n_gwf_points * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_gwf_y, h_gwf_y, opt->n_acq*opt->n_gwf_points * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_gwf_z, h_gwf_z, opt->n_acq*opt->n_gwf_points * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_opt, opt, sizeof(options), hipMemcpyHostToDevice); //copy updated opt to device
		}
	}
	//__________________________________________________________________
	
	//SIMULATION SUBSTRATE
	//__________________________________________________________________
	//substrate
	if (PROCEED){
	h_w = (world*)malloc(sizeof(world));
	//Load info about the substrate so we know how much memory to allocate for it
	PROCEED = get_substrate_info(opt, h_w, sub_fn);}
	
	//allocate substrate memory on host
	if (PROCEED)
	{
	h_centre_x = (float*)malloc(h_w->num_cells * sizeof(float));
	h_centre_y = (float*)malloc(h_w->num_cells * sizeof(float));
	h_radii = (float*)malloc(h_w->num_cells * sizeof(float));
	h_table = (long long*)malloc(h_w->num_voxels * sizeof(long long));
	h_cell_idx = (long long*)malloc(h_w->num_voxels * sizeof(long long));
	
	//load the substrate
	PROCEED = load_substrate(h_centre_x, h_centre_y, h_radii, h_table, h_cell_idx, h_w, sub_fn);
	}

	if (PROCEED)
	{
	//allocate substrate memory on device and check if we stil have space
	hipMalloc(&dev_centre_x, h_w->num_cells * sizeof(float));
	hipMalloc(&dev_centre_y, h_w->num_cells * sizeof(float));
	hipMalloc(&dev_radii, h_w->num_cells * sizeof(float));
	hipMalloc(&dev_cell_idx, h_w->num_voxels * sizeof(long long));
	hipMalloc(&dev_w, sizeof(world));
	error = hipMalloc(&dev_table, h_w->num_voxels * sizeof(long long));
	if (error != hipSuccess)
	{
		std::cout << "\n ERROR: FAILED TO ALLOCATE SUBSTRATE MEMORY ON GPU! \n" << std::endl;
		PROCEED = false;//throw error;
	}
	else {	
		//copy the substrate to the GPU 
		hipMemcpy(dev_table, h_table, h_w->num_voxels * sizeof(long long), hipMemcpyHostToDevice);
		hipMemcpy(dev_w, h_w, sizeof(world), hipMemcpyHostToDevice);
		hipMemcpy(dev_centre_x, h_centre_x, h_w->num_cells * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_centre_y, h_centre_y, h_w->num_cells * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_radii, h_radii, h_w->num_cells * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_cell_idx, h_cell_idx, h_w->num_voxels * sizeof(long long), hipMemcpyHostToDevice);
		}
	}
	//__________________________________________________________________
	
	//SIGNALS AND PHASES
	//__________________________________________________________________
	//allocate them on host
	if (PROCEED)
	{
	h_signal = (float*)malloc(opt->n_acq * sizeof(float));
	h_phase = (float*)malloc(opt->n_acq*opt->Npart * sizeof(float));
	//initialise phase array, it's important that it contains only zeros
	for (long c = 0; c < opt->n_acq*opt->Npart; c++) {h_phase[c] = 0;}
	//allocate memory for arrays on device
	error = hipMalloc(&dev_phase, opt->n_acq*opt->Npart * sizeof(float));
	
	if (error != hipSuccess)
	{
		std::cout << "\n ERROR: FAILED TO ALLOCATE PHASE MEMORY ON GPU! \n" << std::endl;
		PROCEED = false;//throw error;
	}
	else {	
		//copy phase array to device
		hipMemcpy(dev_phase, h_phase, opt->n_acq*opt->Npart * sizeof(float), hipMemcpyHostToDevice);
		}
	}
	//__________________________________________________________________


	//FINAL POSITIONS AND TRANSITION HISTORY
	//__________________________________________________________________
	if (PROCEED)
	{
	//allocate them on host
	h_loc = (int*)malloc(opt->Npart* sizeof(int));
	h_x = (float*)malloc(opt->Npart * sizeof(float));
	h_y = (float*)malloc(opt->Npart * sizeof(float));
	h_z = (float*)malloc(opt->Npart * sizeof(float));

	//allocate memory for arrays on device
	hipMalloc(&dev_loc, opt->Npart * sizeof(int));
	hipMalloc(&dev_x, opt->Npart * sizeof(float));
	hipMalloc(&dev_y, opt->Npart * sizeof(float));
	error = hipMalloc(&dev_z, opt->Npart * sizeof(float));

	if (error != hipSuccess)
	{
		std::cout << "\n ERROR: FAILED TO ALLOCATE TRAJECTORY MEMORY ON GPU. \n" << std::endl;
		PROCEED = false;//throw error;
	}
	else {
			//copy x,y,z  and id arrays to device
		hipMemcpy(dev_loc, h_loc, opt->Npart * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_x, h_x, opt->Npart * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_y, h_y, opt->Npart * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_z, h_z, opt->Npart * sizeof(float), hipMemcpyHostToDevice);
		}	
	}
	//__________________________________________________________________


	//DEFINE GRID TEXTURE
	//__________________________________________________________________
	int blockSize;
	64 > opt->Npart ? blockSize = (int)opt->Npart : blockSize = 64;
	int numBlocks = (int)(opt->Npart + blockSize - 1) / blockSize; //make sure to round up in case N is not an integer multiple of blockSize
	
	//allocate hiprandState for every CUDA thread on the host
	error = hipMalloc(&dev_states, blockSize * numBlocks * sizeof(hiprandState));
	if (error != hipSuccess)
	{
		std::cout << "\n ERROR: FAILED TO ALLOCATE RNG STATES MEMORY ON GPU. \n" << std::endl;
		PROCEED = false;//throw error;
	}
	//__________________________________________________________________
	
	
	//BEGIN SIMULATION
	//__________________________________________________________________
	if (PROCEED)
	{
	std::cout << "Generating GPU grid texture.." << std::endl;
	std::cout << "Number of blocks = " << numBlocks << " AND block size = " << blockSize << std::endl;
	
	std::cout << "Generating initial spin distribution..." << std::endl;
	
	//initialise RNG for all threads
	random_init << < numBlocks, blockSize >> > (dev_states);
	//generate initial particle distribution
	generate_initial_distribution << < numBlocks, blockSize >> > (dev_x, dev_y, dev_z, dev_loc, dev_centre_x, dev_centre_y, dev_radii, dev_table, dev_cell_idx, dev_states, dev_opt, dev_w);
	hipDeviceSynchronize(); //Tell CPU to wait until kernel is done before accessing results. This is necessary because
							//cuda kernel launches do not block the calling CPU thread.
	//launch simulation engine
	std::cout << "Running simulation..." << std::endl;
	engine << < numBlocks, blockSize >> > (dev_x, dev_y, dev_z, dev_loc, dev_centre_x, dev_centre_y, dev_radii, dev_table, dev_cell_idx, dev_states, dev_opt, dev_w, dev_phase,
	dev_gwf_x, dev_gwf_y, dev_gwf_z);
	hipDeviceSynchronize(); 
	//__________________________________________________________________
	
	//DOWNLOAD AND SAVE RESULTS
	//__________________________________________________________________
	//copy phase array back to host machine
	hipMemcpy(h_phase, dev_phase, opt->n_acq*opt->Npart * sizeof(float), hipMemcpyDeviceToHost);
	//copy final particle positions for diagnostics
	hipMemcpy(h_x, dev_x, opt->Npart * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_y, dev_y, opt->Npart * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_z, dev_z, opt->Npart * sizeof(float), hipMemcpyDeviceToHost);

	convert_phase_to_signal(h_phase, h_signal, opt);
	//write results to binary files
	if (opt->save_phase) {save_phase(h_phase, opt, phase_fn);}
	save_signal(h_signal, sig_fn, opt);
	if (opt->save_positions) {save_trajectory(h_x, h_y, h_z, pos_fn, opt);};
	if (opt->save_states) {save_state_history(h_loc, sta_fn, opt);};
	//__________________________________________________________________
	
	duration = (std::clock() - start) / (float)CLOCKS_PER_SEC;
	std::cout << "Elapsed time is: " << duration << " seconds." << std::endl;
	}
	else
	{
		std::cout << "\n FAILED TO LAUNCH SIMULATION ENGINE. ABORTING. \n" << std::endl;
	}

	//FREE RESOURCES
	//__________________________________________________________________
	// Free memory on host
	free(h_signal);
	free(h_phase);
	free(h_gwf_x);
	free(h_gwf_y);
	free(h_gwf_z);
	free(h_x);
	free(h_y);
	free(h_z);
	free(h_w);
	free(opt);
	free(h_loc);
	free(h_centre_x);
	free(h_centre_y);
	free(h_radii);
	free(h_table);
	free(h_cell_idx);
	//free memory on device
	hipFree(dev_phase);
	hipFree(dev_gwf_x);
	hipFree(dev_gwf_y);
	hipFree(dev_gwf_z);
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	hipFree(dev_states);
	hipFree(dev_loc);
	hipFree(dev_w);
	hipFree(dev_opt);
	hipFree(dev_centre_x);
	hipFree(dev_centre_y);
	hipFree(dev_radii);
	hipFree(dev_table);
	hipFree(dev_cell_idx);
	//__________________________________________________________________
	return 0;
}
